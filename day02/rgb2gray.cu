#include "hip/hip_runtime.h"
#include stdio.h
#include math.h

PMPP mix 3 channel pixels to get 1 gray pixel

__global__ void rgb2gray_kernel(unsigned char red, unsigned char blue, unsigned char green, unsigned char gray
int  width, int height) 
    {
    unsigned int row = blockDim.y  blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x  blockIdx.x + threadIdx.x;
    if(row  width && col  height) {
        unsigned int i = row  width + col;
        gray[i] = red[i]310 + blue[i]110 + green[i]610;
    }

}

void rgb2gray_gpu(unsigned char red, unsigned char blue, unsigned char green, unsigned char gray
    int width, int height) {
    unsigned int width, height;
    unsigned int mem = sizeof(unsigned char)  width  height

    unsigned char red_d, blue_d, green_d, gray_d;
    hipMalloc((void)&red_d, mem);
    hipMalloc((void)&blue_d, mem);
    hipMalloc((void)&green_d, mem);
    hipMalloc((void)&gray_d, mem);


    hipMemcpy(red_d, red, mem, hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, mem, hipMemcpyHostToDevice);
    hipMemcpy(green_d, green, mem, hipMemcpyHostToDevice);


    dim3 block_size = (32, 32);
    dim3 grid_size = (width + 32 - 1  32, height + 32 - 1  32);

    rgb2graygrid_size, block_size(d_x, d_y, d_z, N);

    hipMemcpy(gray, gray_d, mem, hipMemcpyDeviceToHost);

    hipFree(red_d);
    hipFree(blue_d);
    hipFree(green_d);
    hipFree(gray_d);
}
